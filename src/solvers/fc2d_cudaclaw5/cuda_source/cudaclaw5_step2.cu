#include "../fc2d_cudaclaw5.h"

#include <fclaw2d_patch.h>
#include <fclaw2d_global.h>
#include <fclaw2d_clawpatch.h>
//#include <fclaw2d_clawpatch.hpp>

#include "../fc2d_cudaclaw5_fort.h"
#include "../fc2d_cudaclaw5_options.h"

#include "cudaclaw5_update_q.h"

double cudaclaw5_step2(fclaw2d_global_t *glob,
                       fclaw2d_patch_t *this_patch,
                       int this_block_idx,
                       int this_patch_idx,
                       double t,
                       double dt)
{
    fc2d_cudaclaw5_vtable_t*  cuclaw5_vt = fc2d_cudaclaw5_vt();
    fc2d_cudaclaw5_options_t* cudaclaw_options;
    int level;
    double *qold, *aux;
    int mx, my, meqn, maux, mbc;
    double xlower, ylower, dx,dy;

    FCLAW_ASSERT(cuclaw5_vt->fort_rpn2 != NULL);
    FCLAW_ASSERT(cuclaw5_vt->fort_rpt2 != NULL);

    cudaclaw_options = fc2d_cudaclaw5_get_options(glob);
    level = this_patch->level;

    fclaw2d_clawpatch_aux_data(glob,this_patch,&aux,&maux);
    fclaw2d_clawpatch_save_current_step(glob, this_patch);
    fclaw2d_clawpatch_grid_data(glob,this_patch,&mx,&my,&mbc,
                                &xlower,&ylower,&dx,&dy);
    fclaw2d_clawpatch_soln_data(glob,this_patch,&qold,&meqn);

    int mwaves = cudaclaw_options->mwaves;
    int maxm = SC_MAX(mx,my);
    double cflgrid = 0.0;

    int mwork = (maxm+2*mbc)*(12*meqn + (meqn+1)*mwaves + 3*maux + 2);
    double* work = new double[mwork];

    int size = meqn*(mx+2*mbc)*(my+2*mbc);
    double* fp = new double[size];
    double* fm = new double[size];
    double* gp = new double[size];
    double* gm = new double[size];

    int ierror = 0;
    cudaclaw5_fort_flux2_t flux2 = CUDACLAW5_FLUX2;

    int* block_corner_count = fclaw2d_patch_block_corner_count(glob,this_patch);


    CUDACLAW5_STEP2(&maxm,&meqn,&maux,&mbc,&mx,&my,qold,aux,
                    &dx,&dy,&dt,&cflgrid,fm,fp,gm,gp,cuclaw5_vt->fort_rpn2,
                    cuclaw5_vt->fort_rpt2,block_corner_count,&ierror);

    /* # update q */
    double dtdx, dtdy;
    dtdx = dt/dx;
    dtdy = dt/dy;

#if 0
    CUDACLAW5_FORT_UPDATE_Q(&meqn,&mx,&my,&mbc,&maux,
                            &dtdx,&dtdy,qold,fp,fm,
                            gp,gm,&cudaclaw_options->mcapa);
#else
  //  cudaclaw5_update_q(meqn,mx,my,mbc,
  //                     dtdx,dtdy,qold,
  //                     fm,fp,gm,gp,cudaclaw_options->mcapa);
    double* qold_dev;
    double* fm_dev;
    double* fp_dev;
    double* gm_dev;
    double* gp_dev;

    fclaw2d_timer_start (&glob->timers[FCLAW2D_TIMER_CUDA_ALLOCATE]);
    hipMalloc((void**)&qold_dev, size * sizeof(double));
    hipMalloc((void**)&fm_dev, size * sizeof(double));
    hipMalloc((void**)&fp_dev, size * sizeof(double));
    hipMalloc((void**)&gm_dev, size * sizeof(double));
    hipMalloc((void**)&gp_dev, size * sizeof(double));
    fclaw2d_timer_stop (&glob->timers[FCLAW2D_TIMER_CUDA_ALLOCATE]);

    hipMemcpy(qold_dev, qold, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fm_dev, fm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fp_dev, fp, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gm_dev, gm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gp_dev, gp, size * sizeof(double), hipMemcpyHostToDevice);

    dim3 dimBlock(mx, my,meqn);
    dim3 dimGrid(1, 1);
    fclaw2d_timer_start (&glob->timers[FCLAW2D_TIMER_CUDA_KERNEL1]);
    cudaclaw5_update_q_cuda<<<dimGrid, dimBlock>>>(mbc, dtdx, dtdy,
                                                   qold_dev, fm_dev, fp_dev,
                                                   gm_dev, gp_dev);
    fclaw2d_timer_stop (&glob->timers[FCLAW2D_TIMER_CUDA_KERNEL1]);
    hipError_t code = hipPeekAtLastError();
    if(code!=hipSuccess){
        printf("ERROR: %s\n",hipGetErrorString(code));
    }

    hipMemcpy(qold, qold_dev, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(qold_dev);
    hipFree(fm_dev);
    hipFree(fp_dev);
    hipFree(gm_dev);
    hipFree(gp_dev);
#endif

    FCLAW_ASSERT(ierror == 0);

    delete [] fp;
    delete [] fm;
    delete [] gp;
    delete [] gm;
    delete [] work;

    return cflgrid;
}

#if 0
/* Use for possible work arrays */
c     # Local variables
      integer i0faddm, i0faddp, i0gaddm, i0gaddp
      integer i0q1d, i0dtdx1, i0dtdy1
      integer i0aux1, i0aux2, i0aux3, i0next, mused, mwork1
      integer i0wave, i0s, i0amdq, i0apdq, i0ql, i0qr, i0auxl
      integer i0auxr

      integer i,j,m

c     Needed by Riemann solvers.  This should be fixed later by a 'context'
c     for a Riemann solver.
      double precision dtcom, dxcom,dycom,tcom
      integer icom, jcom
      common/comxyt/dtcom,dxcom,dycom,tcom,icom,jcomdouble dtdx, double dtdy,
                            double* qold,
                            double* fm, double* fp,
                            double* gm, double* gp);

c     # This should be set to actual time, in case the user wants it
c     # it for some reason in the Riemann solver.

c     # Set up work arrays (these are not used yet)

      i0faddm = 1
      i0faddp = i0faddm +   (maxm+2*mbc)*meqn
      i0gaddm = i0faddp +   (maxm+2*mbc)*meqn
      i0gaddp = i0gaddm + 2*(maxm+2*mbc)*meqn
      i0q1d   = i0gaddp + 2*(maxm+2*mbc)*meqn
      i0dtdx1 = i0q1d   +   (maxm+2*mbc)*meqn
      i0dtdy1 = i0dtdx1 +   (maxm+2*mbc)
      i0aux1  = i0dtdy1 +   (maxm+2*mbc)
      i0aux2  = i0aux1  +   (maxm+2*mbc)*maux
      i0aux3  = i0aux2  +   (maxm+2*mbc)*maux
c
c
      i0next  = i0aux3  + (maxm+2*mbc)*maux    !# next free space
      mused   = i0next - 1                    !# space already used
      mwork1  = mwork - mused           !# remaining space (passed to step2)

      if (mused.gt.mwork) then
         ierror = 1
         return
      endifid need for
c     # global array
c      call cudaclaw5_step2(maxm,maxmx,maxmy,meqn,maux, mbc,
c     &      mx,my, qold,aux,dx,dy,dt,
c     &      cfl,fm,fp,gm,gp,
c     &      work(i0faddm),work(i0faddp),
c     &      work(i0gaddm),work(i0gaddp),
c     &      work(i0q1d),work(i0dtdx1),work(i0dtdy1),
c     &      work(i0aux1),work(i0aux2),work(i0aux3),
c     &      work(i0next),mwork1,rpn2,rpt2,flux2,
c     &      mwaves,mcapa,method,mthlim,block_corner_count,ierror)
#endif

