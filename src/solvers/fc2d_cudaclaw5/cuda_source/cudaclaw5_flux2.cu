#include "hip/hip_runtime.h"
#include "cudaclaw5_flux2.h"
#include <fclaw_base.h>  /* Needed for SC_MIN, SC_MAX */

#include <math.h>


/* Use this version (in swirl example) to test performance hit with function
   pointers */
__device__ void rpn2adv(int idir, int meqn, int mwaves, 
                        int maux, double ql[], double qr[], 
                        double auxl[], double auxr[],
                        double wave[], double s[], 
                        double amdq[], double apdq[])
{
    /* Solve q_t + D q_x = 0, where D = diag([u,u,...,u]), D in R^{meqn x meqn} */
    int mq;

    for(mq = 0; mq < meqn; mq++)
    {
        wave[mq] = qr[mq] - ql[mq];        
    }

    s[0] = auxr[idir];    /* Assume all waves move at the same speed */

    for(mq = 0; mq < meqn; mq++)
    {
        amdq[mq] = SC_MIN(s[0], 0) * wave[mq];
        apdq[mq] = SC_MAX(s[0], 0) * wave[mq];            
    }
}

#if 0
__device__ void rpn2adv_cuda2(int idir, int meqn, int mwaves, int maux,
     double ql[], double qr[], double auxl[], double auxr[],
     double wave[], double s[], double amdq[], double apdq[])
{
    /* wave[mwaves][meqn] */
    /* idir in 0,1 : needed to get correct  */

    wave[0] = qr[0] - ql[0];
    s[0] = auxr[idir];
    amdq[0] = SC_MIN(auxr[idir], 0) * wave[0];
    apdq[0] = SC_MAX(auxr[idir], 0) * wave[0];
}
#endif


#define MEQN   10
#define MAUX   20 
#define MWAVES 10

extern "C"
{

__host__ int cudaclaw5_check_dims(int meqn, int maux, int mwaves)
{
    int check;
    check = (meqn <= MEQN) && (maux <= MAUX) && (mwaves <= MWAVES);
    return check;
}
}


__global__ void cudaclaw5_flux2(int idir, int mx, int my, int meqn, int mbc,
                                int maux, double* qold, double* aux, double dx,
                                double dy, double dt, double* cflgrid,
                                double* fm, double* fp, double* gm, double* gp,
                                double* waves, double *speeds,
                                cudaclaw5_cuda_rpn2_t rpn2, void* rpt2,
                                int mwaves) 
{
    int mq, mw, m;
    int x_stride_q, y_stride_q, I_q;
    int x_stride_aux, y_stride_aux, I_aux;
    int x_stride_waves, y_stride_waves, I_waves;
    int x_stride_s, y_stride_s, I_speeds;

    /* Static memory seems much faster than dynamic memory */
    double ql[MEQN];
    double qr[MEQN];
    double auxl[MAUX];
    double auxr[MAUX];
    double s[MWAVES];
    double wave[MEQN*MWAVES];
    double amdq[MEQN];
    double apdq[MEQN];

    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    /* Array of structures */
    x_stride_q = meqn;
    y_stride_q = (2 * mbc + mx) * x_stride_q;
    I_q = (ix + mbc-1) * x_stride_q + (iy + mbc-1) * y_stride_q;

    x_stride_aux = maux;
    y_stride_aux = (2 * mbc + mx) * x_stride_aux;
    I_aux = (ix + mbc-1) * x_stride_aux + (iy + mbc-1) * y_stride_aux;

    x_stride_waves = mwaves*meqn;
    y_stride_waves = (2 * mbc + mx) * x_stride_waves;
    I_waves = (ix + mbc-1) * x_stride_waves + (iy + mbc-1) * y_stride_waves;

    x_stride_s = mwaves;
    y_stride_s = (2 * mbc + mx) * x_stride_s;
    I_speeds = (ix + mbc-1) * x_stride_s + (iy + mbc-1) * y_stride_s;


    if (idir == 0 && (ix < mx + 2*mbc-1 && iy < my + 2*(mbc-1)))
    {
        for(mq = 0; mq < meqn; mq++)
        {
            ql[mq] = qold[I_q - x_stride_q + mq];
            qr[mq] = qold[I_q + mq];            
        }
        for(m = 0; m < maux; m++)
        {
            auxl[m] = aux[I_aux - x_stride_aux + m];
            auxr[m] = aux[I_aux+m];
        }

        //rpn2adv_cuda2(0, meqn, mwaves, maux, ql, qr, auxl, auxr, wave, s, amdq, apdq);
        rpn2(0, meqn, mwaves, maux, ql, qr, auxl, auxr, wave, s, amdq, apdq);

        for (mq = 0; mq < meqn; mq++) 
        {
            int i = I_q + mq;
            fp[i] = -apdq[mq]; 
            fm[i] = amdq[mq];
        }
    }
    else if (idir == 1 && (ix < mx + 2*(mbc-1) && iy < my + 2*mbc-1))
    {
        for(mq = 0; mq < meqn; mq++)
        {
            ql[mq] = qold[I_q - y_stride_q + mq];
            qr[mq] = qold[I_q + mq];            
        }
        for(m = 0; m < maux; m++)
        {
            auxl[m] = aux[I_aux - y_stride_aux + m];
            auxr[m] = aux[I_aux + m];
        }

        //rpn2adv_cuda2(1, meqn, mwaves, maux, ql, qr, auxl, auxr, wave, s, amdq, apdq);
        rpn2(1, meqn, mwaves, maux, ql, qr, auxl, auxr, wave, s, amdq, apdq);

        for (mq = 0; mq < meqn; mq++) 
        {
            int i = I_q + mq;
            gp[i] = -apdq[mq]; 
            gm[i] = amdq[mq];
        }
    }

    /* Assumes array of structures */
    for (m = 0; m < meqn*mwaves; m++)
    {
        int i = I_waves + m;
        waves[i] = wave[m];
    }

    for (mw = 0; mw < mwaves; mw++)
    {
        int i = I_speeds + mw;
        speeds[i] = s[mw];
    }
}

__global__ void cudaclaw5_compute_cfl(int idir, int mx, int my, int meqn, int mwaves, 
                                      int mbc, double dx, double dy, double dt, 
                                      double *speeds, double* cflgrid)
{
#if 0    
      # from fortran_source/cudaclaw5_flux2.f */

c     # compute maximum wave speed for checking Courant number:
      cfl1d = 0.d0
      do 50 mw=1,mwaves
         do 50 i=1,mx+1
c          # if s>0 use dtdx1d(i) to compute CFL,
c          # if s<0 use dtdx1d(i-1) to compute CFL:
            cfl1d = dmax1(cfl1d, dtdx1d(i)*s(mw,i),
     &                          -dtdx1d(i-1)*s(mw,i))
   50       continue
#endif   
    /* Compute largest waves speeds, scaled by dt/dx,  on grid */


}


__device__ void cudaclaw5_second_order(int idir, int mx, int my, int meqn, int mbc,
                                       int maux, double* qold, double* aux, double dx,
                                       double dy, double dt, double* cflgrid,
                                       double* fm, double* fp, double* gm, double* gp,
                                       double* waves, double *speeds,
                                       cudaclaw5_cuda_rpn2_t rpn2, void* rpt2,
                                       int mwaves) 
{    
    int mq, mw, m;

    /* TODO : Limit waves here */


    /* Compute second order corrections */
    double dtdx = dt/dx;
    for(mq = 0; mq < meqn; mq++)
    {
        double cqxx = 0;
        for(mw = 0; mw < mwaves; mw++)
        {
            m = mw*meqn + mq;
            cqxx += fabs(speeds[mw])*(1.0 - fabs(speeds[mw])*dtdx)*waves[m];
        }
    }
}

