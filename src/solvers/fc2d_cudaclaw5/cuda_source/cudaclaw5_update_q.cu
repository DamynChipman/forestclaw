#include "hip/hip_runtime.h"
#include "cudaclaw5_update_q.h"

__global__ void cudaclaw5_update_q_cuda(int mbc, 
                                        double dtdx, double dtdy,
                                        double* qold, 
                                        double* fm, double* fp, 
                                        double* gm, double* gp)
{
    int meqn = threadIdx.z;
    int x = threadIdx.x;
    int x_stride = blockDim.z;
    int y = threadIdx.y;
    int y_stride = (blockDim.x+2*mbc)*x_stride;
    int i = meqn + (x+mbc)*x_stride + (y+mbc)*y_stride;
    qold[i] = qold[i] - dtdx * (fm[i+x_stride] - fp[i]) 
                      - dtdy * (gm[i+y_stride] - gp[i]);
}


void cudaclaw5_update_q(int meqn, int mx, int my, int mbc, 
                        double dtdx, double dtdy, double qold[], 
                        double fm[], double fp[], 
                        double gm[], double gp[], int mcapa) 
{
    int size = meqn * (mx + 2 * mbc) * (my + 2 * mbc);

    double* qold_dev;
    double* fm_dev;
    double* fp_dev;
    double* gm_dev;
    double* gp_dev;    

    hipMalloc((void**)&qold_dev, size * sizeof(double));
    hipMalloc((void**)&fm_dev, size * sizeof(double));
    hipMalloc((void**)&fp_dev, size * sizeof(double));
    hipMalloc((void**)&gm_dev, size * sizeof(double));
    hipMalloc((void**)&gp_dev, size * sizeof(double));

    hipMemcpy(qold_dev, qold, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fm_dev, fm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fp_dev, fp, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gm_dev, gm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gp_dev, gp, size * sizeof(double), hipMemcpyHostToDevice);

    // this is not optimal
    dim3 dimGrid(mx, my, meqn);
    dim3 dimBlock(1, 1);
    cudaclaw5_update_q_cuda<<<dimBlock, dimGrid>>>(mbc, dtdx, dtdy, 
                                                   qold_dev, fm_dev, fp_dev, 
                                                   gm_dev, gp_dev);
    //equivalent c loop
    /*
    int x_stride = meqn;
    int y_stride = (mx + 2 * mbc)*x_stride;
    for(int m=0;m<meqn;m++){
        for(int x=0;x<mx;x++){
            for(int y=0;y<my;y++){
                int i = m+(x+mbc)*x_stride+(y+mbc)*y_stride;
                qold[i] =qold[i] -dtdx * (fm[i+x_stride] - fp[i]) -
                                  dtdy * (gm[i+y_stride] - gp[i]);

            }
        }
    }
    */

	hipError_t code = hipPeekAtLastError();
    if(code!=hipSuccess){
        printf("ERROR: %s\n",hipGetErrorString(code));
    }

    hipMemcpy(qold, qold_dev, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(qold_dev);
    hipFree(fm_dev);
    hipFree(fp_dev);
    hipFree(gm_dev);
    hipFree(gp_dev);
}
