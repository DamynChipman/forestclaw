#include "hip/hip_runtime.h"
#include "cudaclaw5_update_q.h"

__global__ void cudaclaw5_update_q_cuda(int mbc, 
                                        double dtdx, double dtdy,
                                        double* qold, 
                                        double* fm, double* fp, 
                                        double* gm, double* gp)
{
    int mq = threadIdx.z;
    int x = threadIdx.x;
    int x_stride = blockDim.z;
    int y = threadIdx.y;
    int y_stride = (blockDim.x + 2*mbc)*x_stride;
    int i = mq + (x+mbc)*x_stride + (y+mbc)*y_stride;
    qold[i] = qold[i] - dtdx * (fm[i+x_stride] - fp[i]) 
                      - dtdy * (gm[i+y_stride] - gp[i]);
}

__global__ void cudaclaw5_update_q_cuda2(int mbc, int mx, int my, int meqn,
                                        double dtdx, double dtdy,
                                        double* qold, 
                                        double* fm, double* fp, 
                                        double* gm, double* gp)
{
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;

    if (ix < mx && iy < my)
    {
        int x_stride = meqn;
        int y_stride = (2*mbc + mx)*x_stride;
        int I = (ix+mbc)*x_stride + (iy+mbc)*y_stride;
        int mq;

        for(mq = 0; mq < meqn; mq++)
        {
            int i = I+mq;
            qold[i] = qold[i] - dtdx * (fm[i+x_stride] - fp[i]) 
                      - dtdy * (gm[i+y_stride] - gp[i]);
        }        
    }
}


#if 0
void cudaclaw5_update_q(int meqn, int mx, int my, int mbc, 
                        double dtdx, double dtdy, double qold[], 
                        double fm[], double fp[], 
                        double gm[], double gp[], int mcapa) 
{
    int size = meqn * (mx + 2 * mbc) * (my + 2 * mbc);

    double* qold_dev;
    double* fm_dev;
    double* fp_dev;
    double* gm_dev;
    double* gp_dev;    

    hipMalloc((void**)&qold_dev, size * sizeof(double));
    hipMalloc((void**)&fm_dev, size * sizeof(double));
    hipMalloc((void**)&fp_dev, size * sizeof(double));
    hipMalloc((void**)&gm_dev, size * sizeof(double));
    hipMalloc((void**)&gp_dev, size * sizeof(double));

    hipMemcpy(qold_dev, qold, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fm_dev, fm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fp_dev, fp, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gm_dev, gm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gp_dev, gp, size * sizeof(double), hipMemcpyHostToDevice);

    // this is not optimal
    dim3 dimGrid(mx, my, meqn);
    dim3 dimBlock(1, 1);
    cudaclaw5_update_q_cuda<<<dimBlock, dimGrid>>>(mbc, dtdx, dtdy, 
                                                   qold_dev, fm_dev, fp_dev, 
                                                   gm_dev, gp_dev);
    //equivalent c loop
    /*
    int x_stride = meqn;
    int y_stride = (mx + 2 * mbc)*x_stride;
    for(int m=0;m<meqn;m++){
        for(int x=0;x<mx;x++){
            for(int y=0;y<my;y++){
                int i = m+(x+mbc)*x_stride+(y+mbc)*y_stride;
                qold[i] =qold[i] -dtdx * (fm[i+x_stride] - fp[i]) -
                                  dtdy * (gm[i+y_stride] - gp[i]);

            }
        }
    }
    */

	hipError_t code = hipPeekAtLastError();
    if(code!=hipSuccess){
        printf("ERROR: %s\n",hipGetErrorString(code));
    }

    hipMemcpy(qold, qold_dev, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(qold_dev);
    hipFree(fm_dev);
    hipFree(fp_dev);
    hipFree(gm_dev);
    hipFree(gp_dev);
}
#endif
