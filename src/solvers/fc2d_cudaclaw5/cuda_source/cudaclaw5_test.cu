#include "hip/hip_runtime.h"
#include "../fc2d_cudaclaw5.h"

__global__ void donothing()
{
    /* Do nothing! */

    return;
}

__device__ int addem( int a, int b ) 
{
    return a + b;
}

__global__ void add( int a, int b, int *c ) 
{
    *c = 5;
}


void cudaclaw5_test()
{
    int a,b,c;
    int *dev_c;

    donothing<<<1,1>>>();

    /* Allocate memory on the device */
    hipMalloc( (void**)&dev_c, sizeof(int));

    a = 2;
    b = 7;
    c=2;
    printf("C : %d\n",c);   
    add<<<1,1>>>(a, b, dev_c );   

    hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("C : %d\n",c);   
 
    printf("Kernel result : %d + %d = %d\n",a,b,c);   

    hipFree( dev_c);

    return;
}
