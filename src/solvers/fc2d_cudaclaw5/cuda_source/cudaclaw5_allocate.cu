#include "cudaclaw5_allocate.h"

#include <fclaw2d_global.h>
#include <fclaw2d_patch.h>
#include <fclaw2d_clawpatch.h>
#include <fclaw_timer.h>



void cudaclaw5_allocate_fluxes(struct fclaw2d_global *glob,
                               struct fclaw2d_patch *patch)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    size_t size = fclaw2d_clawpatch_size(glob);

    cudaclaw5_fluxes_t *fluxes = FCLAW_ALLOC(cudaclaw5_fluxes,1);
    fluxes->num_bytes = size*sizeof(double);

    fclaw2d_timer_start (&glob->timers[FCLAW2D_TIMER_EXTRA2]);       
    hipEventRecord(start);
    hipMalloc((void**)&fluxes->qold_dev, fluxes->num_bytes);
    hipMalloc((void**)&fluxes->fm_dev,   fluxes->num_bytes);
    hipMalloc((void**)&fluxes->fp_dev,   fluxes->num_bytes);
    hipMalloc((void**)&fluxes->gm_dev,   fluxes->num_bytes);
    hipMalloc((void**)&fluxes->gp_dev,   fluxes->num_bytes);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    glob->timers[FCLAW2D_TIMER_CUDA_ALLOCATE].cumulative += milliseconds*1e-3;
    fclaw2d_timer_stop (&glob->timers[FCLAW2D_TIMER_EXTRA2]);    

    fclaw2d_patch_set_user_data(glob,patch,fluxes);
}

void cudaclaw5_deallocate_fluxes(fclaw2d_global_t *glob,
                                 fclaw2d_patch_t *patch)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    cudaclaw5_fluxes_t *fluxes = (cudaclaw5_fluxes_t*) 
               fclaw2d_patch_get_user_data(glob,patch);

    FCLAW_ASSERT(fluxes != NULL);

    fclaw2d_timer_start (&glob->timers[FCLAW2D_TIMER_EXTRA2]);       
    hipEventRecord(start);
    hipFree(fluxes->qold_dev);
    hipFree(fluxes->fm_dev);
    hipFree(fluxes->fp_dev);
    hipFree(fluxes->gm_dev);
    hipFree(fluxes->gp_dev);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    glob->timers[FCLAW2D_TIMER_CUDA_ALLOCATE].cumulative += milliseconds*1e-3;
    fclaw2d_timer_stop (&glob->timers[FCLAW2D_TIMER_EXTRA2]);    

    FCLAW_FREE((void*) fluxes);
}

