#include "cudaclaw5_allocate.h"

#include <fclaw2d_global.h>
#include <fclaw2d_patch.h>
#include <fclaw2d_clawpatch.h>
#include <fclaw_timer.h>



void cudaclaw5_allocate_fluxes(struct fclaw2d_global *glob,
                               struct fclaw2d_patch *patch)
{
    size_t size = fclaw2d_clawpatch_size(glob);

    cudaclaw5_fluxes_t *fluxes = FCLAW_ALLOC(cudaclaw5_fluxes,1);
    fluxes->num_bytes = size*sizeof(double);

    /* CPU memory allocation */
    fluxes->fp = new double[size];
    fluxes->fm = new double[size];
    fluxes->gp = new double[size];
    fluxes->gm = new double[size];


    fclaw2d_timer_start (&glob->timers[FCLAW2D_TIMER_CUDA_ALLOCATE]);
    hipMalloc((void**)&fluxes->qold_dev, size * sizeof(double));
    hipMalloc((void**)&fluxes->fm_dev, size * sizeof(double));
    hipMalloc((void**)&fluxes->fp_dev, size * sizeof(double));
    hipMalloc((void**)&fluxes->gm_dev, size * sizeof(double));
    hipMalloc((void**)&fluxes->gp_dev, size * sizeof(double));
    fclaw2d_timer_stop (&glob->timers[FCLAW2D_TIMER_CUDA_ALLOCATE]);

    fclaw2d_patch_set_user_data(glob,patch,fluxes);
}
