#include "../fc2d_cudaclaw.h"
#include "cudaclaw_allocate.h"
#include "cudaclaw_update_q.h"
#include "cudaclaw_flux2.h"

#include <fc2d_cudaclaw_options.h>

#include <fclaw2d_patch.h>
#include <fclaw2d_global.h>
#include <fclaw2d_clawpatch.h>
#include <fclaw2d_clawpatch_options.h>

#include "../fc2d_cudaclaw_check.cu"  /* CHECK defined here */

#include <hipblas.h>
#include <fc2d_cuda_profiler.h>
#include <hipcub/hipcub.hpp>

    
double cudaclaw_step2_batch(fclaw2d_global_t *glob,
        cudaclaw_fluxes_t* array_fluxes_struct, 
        int batch_size, double t, double dt)
{
    PROFILE_CUDA_GROUP("cudaclaw_step2_batch",5);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;
    int i;

    double maxcfl = 0.0;
    //double dtdx, dtdy, s;

    FCLAW_ASSERT(batch_size !=0);

    /* To get patch-independent parameters */
    fc2d_cudaclaw_options_t *clawopt;
    fclaw2d_clawpatch_options_t *clawpatch_opt;

    clawopt = fc2d_cudaclaw_get_options(glob);
    int mwaves = clawopt->mwaves;

    fc2d_cudaclaw_vtable_t*  cuclaw_vt = fc2d_cudaclaw_vt();
    FCLAW_ASSERT(cuclaw_vt->cuda_rpn2 != NULL);


    clawpatch_opt = fclaw2d_clawpatch_get_options(glob);
    int mx = clawpatch_opt->mx;
    int my = clawpatch_opt->my;
    int mbc = clawpatch_opt->mbc;
    int maux = clawpatch_opt->maux;
    int meqn = clawpatch_opt->meqn;  

    cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[0]);
    size_t size = batch_size*(fluxes->num + fluxes->num_aux);
    size_t bytes = size*sizeof(double);
    double *membuffer;
    double* membuffer_dev;

    /* ---------------------------------- Merge Memory ---------------------------------*/ 
    {
        PROFILE_CUDA_GROUP("cudaclaw_copy_loop",7);    
        CHECK(hipHostMalloc((void**)&membuffer,bytes));

        CHECK(hipMalloc((void**)&membuffer_dev, bytes));

        for(i = 0; i < batch_size; i++)   
        {
            cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[i]);    

            int I_q = i*fluxes->num;
            int I_aux = batch_size*fluxes->num + i*fluxes->num_aux;

            memcpy(&membuffer[I_q]  ,fluxes->qold ,fluxes->num_bytes);
            memcpy(&membuffer[I_aux],fluxes->aux  ,fluxes->num_bytes_aux);

            /* Assign gpu pointers */
            fluxes->qold_dev = &membuffer_dev[I_q];
            fluxes->aux_dev  = &membuffer_dev[I_aux];
        }        

        CHECK(hipMemcpy(membuffer_dev, membuffer, bytes, hipMemcpyHostToDevice));
    }        

    /* -------------------------------- Work with array --------------------------------*/ 

    cudaclaw_fluxes_t* array_fluxes_struct_dev = NULL;
    CHECK(hipMalloc(&array_fluxes_struct_dev, batch_size*sizeof(cudaclaw_fluxes_t)));

    CHECK(hipMemcpy(array_fluxes_struct_dev, array_fluxes_struct, 
                     batch_size*sizeof(cudaclaw_fluxes_t), 
                     hipMemcpyHostToDevice));

    dim3 block(128,1,1);
    dim3 grid(1,1,batch_size);

    size_t bytes_per_thread = sizeof(double)*(5*meqn+3*maux+mwaves+meqn*mwaves);
    
    double* maxcflblocks_dev;
    CHECK(hipMalloc(&maxcflblocks_dev,batch_size*sizeof(double))); 
    cudaclaw_flux2_and_update_batch<<<grid,block,128*bytes_per_thread >>>(mx,my,meqn,
                                                                     mbc,maux,mwaves,dt,t,
                                                                     array_fluxes_struct_dev,
								                                     maxcflblocks_dev,
                                                                     cuclaw_vt->cuda_rpn2,
                                                                     cuclaw_vt->cuda_b4step2);
    hipDeviceSynchronize();
    CHECK(hipPeekAtLastError());
	
    /* -------------------------------- Finish CFL ------------------------------------*/ 
    {
        PROFILE_CUDA_GROUP("Finish CFL",1);
        void    *temp_storage_dev = NULL;
        size_t  temp_storage_bytes = 0;
        double  *cflgrid_dev;

        hipMalloc(&cflgrid_dev, sizeof(double));  
        CubDebugExit(hipcub::DeviceReduce::Max(temp_storage_dev,temp_storage_bytes,
                                            maxcflblocks_dev,cflgrid_dev,batch_size));
        hipMalloc(&temp_storage_dev, temp_storage_bytes);
        CubDebugExit(hipcub::DeviceReduce::Max(temp_storage_dev,temp_storage_bytes,
                                            maxcflblocks_dev,cflgrid_dev,batch_size));
        hipMemcpy(&maxcfl, cflgrid_dev, sizeof(double),hipMemcpyDeviceToHost);
        hipFree(temp_storage_dev);
        hipFree(cflgrid_dev);
    }
    /* ------------------------------ Done with CFL ------------------------------------*/ 

    /* -------------------------- Copy q back to host ----------------------------------*/ 
    CHECK(hipMemcpy(membuffer, membuffer_dev, batch_size*fluxes->num_bytes, 
                     hipMemcpyDeviceToHost));

    {
        PROFILE_CUDA_GROUP("Copy back to patches loop",2);
        for (int i = 0; i < batch_size; ++i)    
        {      

            cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[i]);
            int I_q = i*fluxes->num;

            memcpy(fluxes->qold,&membuffer[I_q],fluxes->num_bytes);
        }        
    }

    /* ------------------------------ Clean up -----------------------------------------*/ 
    hipFree(array_fluxes_struct_dev);
    hipFree(membuffer_dev);
    hipHostFree(membuffer);

    return maxcfl;
}

