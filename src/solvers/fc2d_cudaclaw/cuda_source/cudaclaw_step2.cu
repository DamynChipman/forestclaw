#include "../fc2d_cudaclaw.h"

#include "cudaclaw_allocate.h"
#include "cudaclaw_flux2.h"

#include <fc2d_cudaclaw_options.h>

#include <fclaw2d_patch.h>
#include <fclaw2d_global.h>
#include <fclaw2d_clawpatch.h>
#include <fclaw2d_clawpatch_options.h>

#include "../fc2d_cudaclaw_check.cu"  /* CHECK defined here */

#include <hipblas.h>
#include <fc2d_cuda_profiler.h>
#include <hipcub/hipcub.hpp>

static double* s_membuffer;
static double* s_membuffer_dev;

cudaclaw_fluxes_t* s_array_fluxes_struct_dev;


void cudaclaw_allocate_buffers(fclaw2d_global_t *glob)
{
    fclaw2d_clawpatch_options_t *clawpatch_opt = fclaw2d_clawpatch_get_options(glob);
    int mx = clawpatch_opt->mx;
    int my = clawpatch_opt->my;
    int mbc = clawpatch_opt->mbc;
    int maux = clawpatch_opt->maux;
    int meqn = clawpatch_opt->meqn;  

    int batch_size = FC2D_CUDACLAW_BUFFER_LEN;
    size_t size = (2*mbc+mx)*(2*mbc+my);
    size_t bytes = batch_size*size*(meqn + maux)*sizeof(double);

    CHECK(hipHostMalloc(&s_membuffer,bytes));    
    CHECK(hipMalloc(&s_membuffer_dev, bytes)); 
    CHECK(hipMalloc(&s_array_fluxes_struct_dev, 
                     batch_size*sizeof(cudaclaw_fluxes_t)));
}

void cudaclaw_deallocate_buffers(fclaw2d_global_t *glob)
{
    hipHostFree(s_membuffer);
    hipFree(s_membuffer_dev);
    hipFree(s_array_fluxes_struct_dev);
}


double cudaclaw_step2_batch(fclaw2d_global_t *glob,
        cudaclaw_fluxes_t* array_fluxes_struct, 
        int batch_size, double t, double dt)
{
    PROFILE_CUDA_GROUP("cudaclaw_step2_batch",5);
    int i;

    double maxcfl = 0.0;

    FCLAW_ASSERT(batch_size !=0);

    /* To get patch-independent parameters */
    fc2d_cudaclaw_options_t *clawopt;
    fclaw2d_clawpatch_options_t *clawpatch_opt;

    clawopt = fc2d_cudaclaw_get_options(glob);
    int mwaves = clawopt->mwaves;

    fc2d_cudaclaw_vtable_t*  cuclaw_vt = fc2d_cudaclaw_vt();
    FCLAW_ASSERT(cuclaw_vt->cuda_rpn2 != NULL);


    clawpatch_opt = fclaw2d_clawpatch_get_options(glob);
    int mx = clawpatch_opt->mx;
    int my = clawpatch_opt->my;
    int mbc = clawpatch_opt->mbc;
    int maux = clawpatch_opt->maux;
    int meqn = clawpatch_opt->meqn;  

    cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[0]);
    size_t size = batch_size*(fluxes->num + fluxes->num_aux);
    size_t bytes = size*sizeof(double);

    /* ---------------------------------- Merge Memory ---------------------------------*/ 
    FCLAW_ASSERT(s_membuffer != NULL);
    FCLAW_ASSERT(s_membuffer_dev != NULL);

    {
        PROFILE_CUDA_GROUP("Copy q and aux to CPU memory buffer",3);    
        for(i = 0; i < batch_size; i++)   
        {
            cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[i]);    

            int I_q = i*fluxes->num;
            memcpy(&s_membuffer[I_q]  ,fluxes->qold ,fluxes->num_bytes);
            fluxes->qold_dev = &s_membuffer_dev[I_q];

            if (fluxes->num_aux > 0)
            {
                int I_aux = batch_size*fluxes->num + i*fluxes->num_aux;
                memcpy(&s_membuffer[I_aux],fluxes->aux  ,fluxes->num_bytes_aux);                
                fluxes->aux_dev  = &s_membuffer_dev[I_aux];
            }
        }   
    }     

    {
        PROFILE_CUDA_GROUP("Copy buffer to device",7);              
        CHECK(hipMemcpy(s_membuffer_dev, s_membuffer, bytes, hipMemcpyHostToDevice));            
    }            


    /* -------------------------------- Work with array --------------------------------*/ 


    FCLAW_ASSERT(s_array_fluxes_struct_dev != NULL);

    CHECK(hipMemcpy(s_array_fluxes_struct_dev, array_fluxes_struct, 
                     batch_size*sizeof(cudaclaw_fluxes_t), 
                     hipMemcpyHostToDevice));


    /* Data needed to reduce CFL number */
    double* maxcflblocks_dev;    
    CHECK(hipMalloc(&maxcflblocks_dev,batch_size*sizeof(double)));         
    

    /* Configure kernel */
    int block_size = 128;
    dim3 block(block_size,1,1);
    dim3 grid(1,1,batch_size);

    /* Determine shared memory size */
    int mwork = 7*meqn+3*maux+mwaves+meqn*mwaves;
    size_t bytes_per_thread = sizeof(double)*mwork;
    bytes = bytes_per_thread*block_size;
    
    cudaclaw_flux2_and_update_batch<<<grid,block,bytes>>>(mx,my,meqn,mbc,maux,mwaves,
                                                          mwork,dt,t,
                                                          s_array_fluxes_struct_dev,
                                                          maxcflblocks_dev,
                                                          cuclaw_vt->cuda_rpn2,
                                                          cuclaw_vt->cuda_b4step2);
    hipDeviceSynchronize();
    CHECK(hipPeekAtLastError());
	
    /* -------------------------------- Finish CFL ------------------------------------*/ 
    {
        PROFILE_CUDA_GROUP("Finish CFL",1);
        void    *temp_storage_dev = NULL;
        size_t  temp_storage_bytes = 0;
        double  *cflgrid_dev;

        hipMalloc(&cflgrid_dev, sizeof(double));  
        CubDebugExit(hipcub::DeviceReduce::Max(temp_storage_dev,temp_storage_bytes,
                                            maxcflblocks_dev,cflgrid_dev,batch_size));
        hipMalloc(&temp_storage_dev, temp_storage_bytes);
        CubDebugExit(hipcub::DeviceReduce::Max(temp_storage_dev,temp_storage_bytes,
                                            maxcflblocks_dev,cflgrid_dev,batch_size));
        hipMemcpy(&maxcfl, cflgrid_dev, sizeof(double),hipMemcpyDeviceToHost);
        hipFree(temp_storage_dev);
        hipFree(cflgrid_dev);
    }

    /* -------------------------- Copy q back to host ----------------------------------*/ 
    CHECK(hipMemcpy(s_membuffer, s_membuffer_dev, batch_size*fluxes->num_bytes, 
                     hipMemcpyDeviceToHost));

    {
        PROFILE_CUDA_GROUP("Copy back to patches loop",2);
        for (int i = 0; i < batch_size; ++i)    
        {      

            cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[i]);
            int I_q = i*fluxes->num;

            memcpy(fluxes->qold,&s_membuffer[I_q],fluxes->num_bytes);
        }        
    }

    return maxcfl;
}

