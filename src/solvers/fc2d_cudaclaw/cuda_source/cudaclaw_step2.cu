#include "hip/hip_runtime.h"
#include "../fc2d_cudaclaw.h"
#include "cudaclaw_allocate.h"

#include <fclaw2d_patch.h>
#include <fclaw2d_global.h>
#include <fclaw2d_clawpatch.h>

#include "../fc2d_cudaclaw_options.h"
#include "src/patches/clawpatch/fclaw2d_clawpatch_options.h"
#include <fclaw2d_global.h>

#include "cudaclaw_update_q.h"
#include "cudaclaw_flux2.h"

#include "../fc2d_cudaclaw_check.cu"  /* CHECK defined here */
#include <hipblas.h>
    
double cudaclaw_step2_batch( fclaw2d_global_t *glob,
        cudaclaw_fluxes_t* array_fluxes_struct, 
        int batch_size, double dt)
{
    double maxcfl = 0.0;

    /* To get patch-independent parameters */
    fc2d_cudaclaw_options_t *clawopt;
    fclaw2d_clawpatch_options_t *clawpatch_opt;

    clawopt = fc2d_cudaclaw_get_options(glob);
    int mwaves = clawopt->mwaves;

    fc2d_cudaclaw_vtable_t*  cuclaw_vt = fc2d_cudaclaw_vt();
    FCLAW_ASSERT(cuclaw_vt->cuda_rpn2 != NULL);


    clawpatch_opt = fclaw2d_clawpatch_get_options(glob);
    int mx = clawpatch_opt->mx;
    int my = clawpatch_opt->my;
    int mbc = clawpatch_opt->mbc;
    int maux = clawpatch_opt->maux;
    int meqn = clawpatch_opt->meqn;

//     cudaclaw_fluxes_t* array_fluxes_struct = (cudaclaw_fluxes_t*)
//         malloc(batch_size*sizeof(cudaclaw_fluxes_t));
    cudaclaw_fluxes_t* array_fluxes_struct_dev = NULL;
    hipMalloc(&array_fluxes_struct_dev, batch_size*sizeof(cudaclaw_fluxes_t));
// 
//     for (int i = 0; i < batch_size; ++i)
//     {
//         array_fluxes_struct[i] = *(array_ptr_fluxes[i]);
//     }
    hipMemcpy(array_fluxes_struct_dev, array_fluxes_struct, batch_size*sizeof(cudaclaw_fluxes_t), hipMemcpyHostToDevice);
    // launch the merged kernel

    dim3 block(128,1,1);
    //int grid = (mx+2*mbc-1)*(my+2*(mbc-1)+block-1)/block;
    dim3 grid(1,1,batch_size);

    size_t bytes_per_thread = sizeof(double)*(5*meqn+3*maux+mwaves+meqn*mwaves);

    cudaclaw_flux2_and_update_batch<<<grid,block,128*bytes_per_thread>>>(mx,my,meqn,
                                                                     mbc,maux,mwaves,dt,
                                                                     array_fluxes_struct_dev,
                                                                     cuclaw_vt->cuda_rpn2);

    hipDeviceSynchronize();
    CHECK(hipPeekAtLastError());


#if 0
    // collect max cfl numbers
    int n = (2*mbc+mx)*(2*mbc+my)*mwaves*2;
    int maxidx;

    // TODO: batch cublas call
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    for (int i = 0; i < batch_size; ++i)
    {
        cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[i]);
        int stat = hipblasIdamax(handle,n,
            fluxes->speeds_dev,1,&maxidx);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
                printf ("hipblasIdamax failed");
                hipblasDestroy(handle);
                return EXIT_FAILURE;
        }
        double maxabsspeed_patch = 0.0;
        double maxcfl_patch = 0.0;
        hipMemcpy(&maxabsspeed_patch,fluxes->speeds_dev+maxidx-1,sizeof(double),hipMemcpyDeviceToHost);
        //cflgrid = maxidx < (2*mbc+mx)*(2*mbc+my) ? maxabsspeed*dt/dx : maxabsspeed*dt/dy;

        // TODO: handle cases where dx != dy
        maxcfl_patch = maxabsspeed_patch*dt/fluxes->dx;
        maxcfl = max(maxcfl_patch,maxcfl);
    }
    hipblasDestroy(handle);
#endif    

//     free(array_fluxes_struct);
    hipFree(array_fluxes_struct_dev);
    return maxcfl;
}

#if 0
double cudaclaw_step2(fclaw2d_global_t *glob,
                      fclaw2d_patch_t *this_patch,
                      int this_block_idx,
                      int this_patch_idx,
                      double t,
                      double dt)
{
<<<<<<< HEAD
    int n, maxidx;
    double maxabsspeed, s;
=======
    int mx, my, meqn, maux, mbc;
    double xlower, ylower, dx,dy;
    double cflgrid, s;

    int maxidx, n;
    double maxabsspeed;
    double dtdx, dtdy;
    double *qold, *aux;
>>>>>>> shawn/batch_step2

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;
    
    fc2d_cudaclaw_vtable_t*  cuclaw_vt = fc2d_cudaclaw_vt();

<<<<<<< HEAD
    double *qold, *aux;
    int mx, my, meqn, maux, mbc;
    double xlower, ylower, dx,dy;
    double cflgrid;
    double dtdx, dtdy;


=======
>>>>>>> shawn/batch_step2
    fc2d_cudaclaw_options_t* cuda_opt = fc2d_cudaclaw_get_options(glob);

    cudaclaw_fluxes_t *fluxes = (cudaclaw_fluxes_t*) 
               fclaw2d_patch_get_user_data(glob,this_patch);

    FCLAW_ASSERT(fluxes != NULL);

    FCLAW_ASSERT(cuclaw_vt->cuda_rpn2 != NULL);
    //FCLAW_ASSERT(cuclaw_vt->cuda_rpt2 != NULL);

    fclaw2d_clawpatch_aux_data(glob,this_patch,&aux,&maux);
    fclaw2d_clawpatch_save_current_step(glob, this_patch);
    fclaw2d_clawpatch_grid_data(glob,this_patch,&mx,&my,&mbc,
                                &xlower,&ylower,&dx,&dy);
    fclaw2d_clawpatch_soln_data(glob,this_patch,&qold,&meqn);

#if 0
    int mwork = (maxm+2*mbc)*(12*meqn + (meqn+1)*mwaves + 3*maux + 2);
    double* work = new double[mwork];
#endif    


    int ierror = 0;
    // cudaclaw_fort_flux2_t flux2 = CUDACLAW_FLUX2;

    int* block_corner_count = fclaw2d_patch_block_corner_count(glob,this_patch);

    size_t size = fclaw2d_clawpatch_size(glob);

    /* -------------------------- Construct fluctuations -------------------------------*/ 
    hipEventRecord(start);

    hipMemcpy(fluxes->qold_dev, qold, fluxes->num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(fluxes->aux_dev, aux, fluxes->num_bytes_aux, hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); 
    glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY].cumulative += milliseconds*1e-3;

    {
<<<<<<< HEAD
        /* ---------------------------------------------------------------------------- */
        /* Update patch */
        /* ---------------------------------------------------------------------------- */

        dim3 block(32*32,1,1);
        dim3 grid((mx+2*mbc-1)*(my+2*(mbc-1)+block.x-1)/block.x,1,1);
=======
        int block = 128;
        //int grid = (mx+2*mbc-1)*(my+2*(mbc-1)+block-1)/block;
        int grid=1;
>>>>>>> shawn/batch_step2

        int mwaves = cuda_opt->mwaves;
        int bytes_per_thread = sizeof(double)*(5*meqn+3*maux+mwaves+meqn*mwaves);

        cflgrid = 0.0;

        dtdx = dt/dx;
        dtdy = dt/dy;

        hipEventRecord(start);

<<<<<<< HEAD
        cudaclaw_flux2<<<grid, block>>>(mx,my,meqn,mbc,maux,mwaves, 
=======
        /* ---------------------------------------------------------------------------- */
        /* X direction */
        /* ---------------------------------------------------------------------------- */
        cudaclaw_flux2<<<grid, block,block*bytes_per_thread>>>(mx,my,meqn,mbc,maux,mwaves, 
                                        dtdx, dtdy,
>>>>>>> shawn/batch_step2
                                        fluxes->qold_dev, fluxes->aux_dev,
                                        fluxes->fm_dev,fluxes->fp_dev,
                                        fluxes->gm_dev,fluxes->gp_dev,
                                        fluxes->waves_dev, fluxes->speeds_dev,
                                        cuclaw_vt->cuda_rpn2);
        CHECK(hipPeekAtLastError());

        hipDeviceSynchronize();

<<<<<<< HEAD
        dtdx = dt/dx;
        dtdy = dt/dy;
#if 0

=======
        /* -------------------------- Compute CFL --------------------------------------*/ 
>>>>>>> shawn/batch_step2
        n = (2*mbc+mx)*(2*mbc+my)*mwaves*2;

        hipblasStatus_t stat;
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        stat = hipblasIdamax(handle,n,fluxes->speeds_dev,1,&maxidx);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
                printf ("hipblasIdamax failed");
                hipblasDestroy(handle);
                return EXIT_FAILURE;
        }
        hipMemcpy(&maxabsspeed,fluxes->speeds_dev+maxidx-1,sizeof(double),hipMemcpyDeviceToHost);
        s = fabs(maxabsspeed);
<<<<<<< HEAD
        cflgrid = maxidx < n/2 ? s*dtdx : s*dtdy;        
        hipblasDestroy(handle);
#endif        
=======
	    cflgrid = maxidx < n/2 ? s*dtdx : s*dtdy;
        hipblasDestroy(handle);
>>>>>>> shawn/batch_step2

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        glob->timers[FCLAW2D_TIMER_CUDA_KERNEL1].cumulative += milliseconds*1e-3;        
    }


#if 0                                               
    /* -------------------------- Update solution --------------------------------------*/ 
    hipEventRecord(start);

    dim3 block(32,32);  
    dim3 grid((mx+block.x-1)/block.x,(my+block.y-1)/block.y);

    cudaclaw_update_q_cuda2<<<grid, block>>>(mbc, mx,my,meqn,dtdx, dtdy, 
                                             fluxes->qold_dev, 
                                             fluxes->fm_dev, fluxes->fp_dev,
                                             fluxes->gm_dev, fluxes->gp_dev);
    CHECK(hipPeekAtLastError());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    glob->timers[FCLAW2D_TIMER_CUDA_KERNEL2].cumulative += milliseconds*1e-3;
#endif                                               

    /* -------------------------- Copy q back to host ----------------------------------*/ 
    hipEventRecord(start);

    hipMemcpy(qold, fluxes->qold_dev, fluxes->num_bytes, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY].cumulative += milliseconds*1e-3;
    
    /* ------------------------------ Clean up -----------------------------------------*/ 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    FCLAW_ASSERT(ierror == 0);

    return cflgrid;
}
#endif

#if 0
/* Use for possible work arrays */
c     # Local variables
      integer i0faddm, i0faddp, i0gaddm, i0gaddp
      integer i0q1d, i0dtdx1, i0dtdy1
      integer i0aux1, i0aux2, i0aux3, i0next, mused, mwork1
      integer i0wave, i0s, i0amdq, i0apdq, i0ql, i0qr, i0auxl
      integer i0auxr

      integer i,j,m

c     Needed by Riemann solvers.  This should be fixed later by a 'context'
c     for a Riemann solver.
      double precision dtcom, dxcom,dycom,tcom
      integer icom, jcom
      common/comxyt/dtcom,dxcom,dycom,tcom,icom,jcomdouble dtdx, double dtdy,
                            double* qold,
                            double* fm, double* fp,
                            double* gm, double* gp);

c     # This should be set to actual time, in case the user wants it
c     # it for some reason in the Riemann solver.

c     # Set up work arrays (these are not used yet)

      i0faddm = 1
      i0faddp = i0faddm +   (maxm+2*mbc)*meqn
      i0gaddm = i0faddp +   (maxm+2*mbc)*meqn
      i0gaddp = i0gaddm + 2*(maxm+2*mbc)*meqn
      i0q1d   = i0gaddp + 2*(maxm+2*mbc)*meqn
      i0dtdx1 = i0q1d   +   (maxm+2*mbc)*meqn
      i0dtdy1 = i0dtdx1 +   (maxm+2*mbc)
      i0aux1  = i0dtdy1 +   (maxm+2*mbc)
      i0aux2  = i0aux1  +   (maxm+2*mbc)*maux
      i0aux3  = i0aux2  +   (maxm+2*mbc)*maux
c
c
      i0next  = i0aux3  + (maxm+2*mbc)*maux    !# next free space
      mused   = i0next - 1                    !# space already used
      mwork1  = mwork - mused           !# remaining space (passed to step2)

      if (mused.gt.mwork) then
         ierror = 1
         return
      endifid need for
c     # global array
c      call cudaclaw_step2(maxm,maxmx,maxmy,meqn,maux, mbc,
c     &      mx,my, qold,aux,dx,dy,dt,
c     &      cfl,fm,fp,gm,gp,
c     &      work(i0faddm),work(i0faddp),
c     &      work(i0gaddm),work(i0gaddp),
c     &      work(i0q1d),work(i0dtdx1),work(i0dtdy1),
c     &      work(i0aux1),work(i0aux2),work(i0aux3),
c     &      work(i0next),mwork1,rpn2,rpt2,flux2,
c     &      mwaves,mcapa,method,mthlim,block_corner_count,ierror)
#endif

