#include "../fc2d_cudaclaw.h"
#include "cudaclaw_allocate.h"

#include <fclaw2d_patch.h>
#include <fclaw2d_global.h>
#include <fclaw2d_clawpatch.h>

#include "../fc2d_cudaclaw_options.h"

#include "cudaclaw_update_q.h"
#include "cudaclaw_flux2.h"

#include "../fc2d_cudaclaw_check.cu"  /* CHECK defined here */

double cudaclaw_step2(fclaw2d_global_t *glob,
                      fclaw2d_patch_t *this_patch,
                      int this_block_idx,
                      int this_patch_idx,
                      double t,
                      double dt)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;
    
    fc2d_cudaclaw_vtable_t*  cuclaw_vt = fc2d_cudaclaw_vt();

    double *qold, *aux;
    int mx, my, meqn, maux, mbc;
    double xlower, ylower, dx,dy;
    double cflgrid;

    fc2d_cudaclaw_options_t* cuda_opt = fc2d_cudaclaw_get_options(glob);

    cudaclaw_fluxes_t *fluxes = (cudaclaw_fluxes_t*) 
               fclaw2d_patch_get_user_data(glob,this_patch);

    FCLAW_ASSERT(fluxes != NULL);

    FCLAW_ASSERT(cuclaw_vt->cuda_rpn2 != NULL);
    //FCLAW_ASSERT(cuclaw_vt->cuda_rpt2 != NULL);

    fclaw2d_clawpatch_aux_data(glob,this_patch,&aux,&maux);
    fclaw2d_clawpatch_save_current_step(glob, this_patch);
    fclaw2d_clawpatch_grid_data(glob,this_patch,&mx,&my,&mbc,
                                &xlower,&ylower,&dx,&dy);
    fclaw2d_clawpatch_soln_data(glob,this_patch,&qold,&meqn);

#if 0
    int mwork = (maxm+2*mbc)*(12*meqn + (meqn+1)*mwaves + 3*maux + 2);
    double* work = new double[mwork];
#endif    


    int ierror = 0;
    // cudaclaw_fort_flux2_t flux2 = CUDACLAW_FLUX2;

    int* block_corner_count = fclaw2d_patch_block_corner_count(glob,this_patch);

    size_t size = fclaw2d_clawpatch_size(glob);

    /* -------------------------- Construct fluctuations -------------------------------*/ 
    hipEventRecord(start);

    CHECK(hipMemcpy(fluxes->qold_dev, qold, fluxes->num_bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(fluxes->aux_dev, aux, fluxes->num_bytes_aux, hipMemcpyHostToDevice));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); 
    glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY].cumulative += milliseconds*1e-3;

    {
        dim3 block(32,32);  
        dim3 grid((mx+2*mbc-1+block.x-1)/block.x,(my+2*(mbc-1)+block.y-1)/block.y);

        int mwaves = cuda_opt->mwaves;
        cflgrid = 0.0;

        hipEventRecord(start);

        /* ---------------------------------------------------------------------------- */
        /* X direction */
        /* ---------------------------------------------------------------------------- */
        cudaclaw_flux2<<<grid, block>>>(0,mx,my,meqn,mbc,maux,mwaves, 
                                        fluxes->qold_dev,
                                        fluxes->aux_dev, dx,dy,dt,&cflgrid,
                                        fluxes->fm_dev,fluxes->fp_dev,
                                        fluxes->gm_dev,fluxes->gp_dev,
                                        fluxes->waves_dev, fluxes->speeds_dev,
                                        cuclaw_vt->cuda_rpn2, NULL);
        CHECK(hipPeekAtLastError());

        hipDeviceSynchronize();

#if 0
        cudaclaw_compute_cfl<<<grid, block>>>(0,mx,my,meqn,mwaves, mbc,
                                               dx,dy,dt,fluxes->speeds_dev, &cflgrid);
#endif                                               

        /* ---------------------------------------------------------------------------- */
        /* Y direction */
        /* ---------------------------------------------------------------------------- */
        cudaclaw_flux2<<<grid, block>>>(1,mx,my,meqn,mbc,maux,mwaves,
                                        fluxes->qold_dev,fluxes->aux_dev, 
                                        dx,dy,dt,&cflgrid,
                                        fluxes->fm_dev,fluxes->fp_dev,
                                        fluxes->gm_dev,fluxes->gp_dev,
                                        fluxes->waves_dev, fluxes->speeds_dev,
                                        cuclaw_vt->cuda_rpn2,NULL);
        CHECK(hipPeekAtLastError());
        hipDeviceSynchronize();

#if 0
        cudaclaw_compute_cfl<<<grid, block>>>(1,mx,my,meqn,mwaves, mbc,
                                               dx,dy,dt,fluxes->speeds_dev, &cflgrid);
#endif                                               

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        glob->timers[FCLAW2D_TIMER_CUDA_KERNEL1].cumulative += milliseconds*1e-3;        
    }


    /* -------------------------- Update solution --------------------------------------*/ 
    double dtdx, dtdy;
    dtdx = dt/dx;
    dtdy = dt/dy;

    hipEventRecord(start);

    dim3 block(32,32);  
    dim3 grid((mx+block.x-1)/block.x,(my+block.y-1)/block.y);

    cudaclaw_update_q_cuda2<<<grid, block>>>(mbc, mx,my,meqn,dtdx, dtdy, 
                                             fluxes->qold_dev, 
                                             fluxes->fm_dev, fluxes->fp_dev,
                                             fluxes->gm_dev, fluxes->gp_dev);
    CHECK(hipPeekAtLastError());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    glob->timers[FCLAW2D_TIMER_CUDA_KERNEL2].cumulative += milliseconds*1e-3;

    /* -------------------------- Copy q back to host ----------------------------------*/ 
    hipEventRecord(start);
    CHECK(hipMemcpy(qold, fluxes->qold_dev, fluxes->num_bytes, hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY].cumulative += milliseconds*1e-3;
    
    /* ------------------------------ Clean up -----------------------------------------*/ 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    FCLAW_ASSERT(ierror == 0);

    return cflgrid;
}

#if 0
/* Use for possible work arrays */
c     # Local variables
      integer i0faddm, i0faddp, i0gaddm, i0gaddp
      integer i0q1d, i0dtdx1, i0dtdy1
      integer i0aux1, i0aux2, i0aux3, i0next, mused, mwork1
      integer i0wave, i0s, i0amdq, i0apdq, i0ql, i0qr, i0auxl
      integer i0auxr

      integer i,j,m

c     Needed by Riemann solvers.  This should be fixed later by a 'context'
c     for a Riemann solver.
      double precision dtcom, dxcom,dycom,tcom
      integer icom, jcom
      common/comxyt/dtcom,dxcom,dycom,tcom,icom,jcomdouble dtdx, double dtdy,
                            double* qold,
                            double* fm, double* fp,
                            double* gm, double* gp);

c     # This should be set to actual time, in case the user wants it
c     # it for some reason in the Riemann solver.

c     # Set up work arrays (these are not used yet)

      i0faddm = 1
      i0faddp = i0faddm +   (maxm+2*mbc)*meqn
      i0gaddm = i0faddp +   (maxm+2*mbc)*meqn
      i0gaddp = i0gaddm + 2*(maxm+2*mbc)*meqn
      i0q1d   = i0gaddp + 2*(maxm+2*mbc)*meqn
      i0dtdx1 = i0q1d   +   (maxm+2*mbc)*meqn
      i0dtdy1 = i0dtdx1 +   (maxm+2*mbc)
      i0aux1  = i0dtdy1 +   (maxm+2*mbc)
      i0aux2  = i0aux1  +   (maxm+2*mbc)*maux
      i0aux3  = i0aux2  +   (maxm+2*mbc)*maux
c
c
      i0next  = i0aux3  + (maxm+2*mbc)*maux    !# next free space
      mused   = i0next - 1                    !# space already used
      mwork1  = mwork - mused           !# remaining space (passed to step2)

      if (mused.gt.mwork) then
         ierror = 1
         return
      endifid need for
c     # global array
c      call cudaclaw_step2(maxm,maxmx,maxmy,meqn,maux, mbc,
c     &      mx,my, qold,aux,dx,dy,dt,
c     &      cfl,fm,fp,gm,gp,
c     &      work(i0faddm),work(i0faddp),
c     &      work(i0gaddm),work(i0gaddp),
c     &      work(i0q1d),work(i0dtdx1),work(i0dtdy1),
c     &      work(i0aux1),work(i0aux2),work(i0aux3),
c     &      work(i0next),mwork1,rpn2,rpt2,flux2,
c     &      mwaves,mcapa,method,mthlim,block_corner_count,ierror)
#endif

