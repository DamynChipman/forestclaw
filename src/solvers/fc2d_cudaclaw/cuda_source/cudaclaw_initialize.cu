/*
  Copyright (c) 2018 Carsten Burstedde, Donna Calhoun, Melody Shih, Scott Aiton, 
  Xinsheng Qin.
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

  * Redistributions of source code must retain the above copyright notice, this
  list of conditions and the following disclaimer.
  * Redistributions in binary form must reproduce the above copyright notice,
  this list of conditions and the following disclaimer in the documentation
  and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
  DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
  FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
  DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
  SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
  OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include "../fc2d_cudaclaw_cuda.h"

#include <fclaw2d_global.h>

void fc2d_cudaclaw_initialize_GPUs(fclaw2d_global_t *glob)
{
    int mpirank, count, device_num;
    hipError_t code;
    
    mpirank = glob->mpirank;

    code = hipGetDeviceCount(&count);
    if (code != hipSuccess) 
    {
        fprintf(stderr,"ERROR : %s\n", hipGetErrorString(code));
        exit(code);
    }

    device_num = mpirank % count;  
    printf("mpirank %d assigned to GPU %d\n",mpirank,device_num); 

    code = hipSetDevice(device_num);
    if (code != hipSuccess) 
    {
        fprintf(stderr,"ERROR : %s\n", hipGetErrorString(code));
        exit(code);
    }

}




