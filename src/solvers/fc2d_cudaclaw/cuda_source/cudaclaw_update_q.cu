#include "hip/hip_runtime.h"
#include "cudaclaw_update_q.h"

#if 0
__global__ void cudaclaw_update_q_cuda(int mbc, 
                                        double dtdx, double dtdy,
                                        double* qold, 
                                        double* fm, double* fp, 
                                        double* gm, double* gp)
{
    int mq = threadIdx.z;
    int x = threadIdx.x;
    int x_stride = blockDim.z;
    int y = threadIdx.y;
    int y_stride = (blockDim.x + 2*mbc)*x_stride;
    int i = mq + (x+mbc)*x_stride + (y+mbc)*y_stride;
    qold[i] = qold[i] - dtdx * (fm[i+x_stride] - fp[i]) 
                      - dtdy * (gm[i+y_stride] - gp[i]);
}
#endif

__global__ void cudaclaw_update_q_cuda2(int mbc, int mx, int my, int meqn,
                                        double dtdx, double dtdy,
                                        double* qold, 
                                        double* fm, double* fp, 
                                        double* gm, double* gp)
{
    int xs,ys,zs;
    int I, I_q;
    int ix, iy;
    int mq;

    ix = threadIdx.x + blockIdx.x*blockDim.x;
    iy = threadIdx.y + blockIdx.y*blockDim.y;

    xs = 1;
    ys = (2*mbc + mx)*xs;
    zs = (2*mbc + my)*ys*xs;

    if (ix < mx && iy < my)
    {
        I = (ix+mbc)*xs + (iy+mbc)*ys;

        for(mq = 0; mq < meqn; mq++)
        {
            I_q = I + mq*zs;
            qold[I_q] = qold[I_q] - dtdx * (fm[I_q + xs] - fp[I_q]) 
                                  - dtdy * (gm[I_q + ys] - gp[I_q]);
        }        
    }
}


#if 0
void cudaclaw_update_q(int meqn, int mx, int my, int mbc, 
                        double dtdx, double dtdy, double qold[], 
                        double fm[], double fp[], 
                        double gm[], double gp[], int mcapa) 
{
    int size = meqn * (mx + 2 * mbc) * (my + 2 * mbc);

    double* qold_dev;
    double* fm_dev;
    double* fp_dev;
    double* gm_dev;
    double* gp_dev;    

    hipMalloc((void**)&qold_dev, size * sizeof(double));
    hipMalloc((void**)&fm_dev, size * sizeof(double));
    hipMalloc((void**)&fp_dev, size * sizeof(double));
    hipMalloc((void**)&gm_dev, size * sizeof(double));
    hipMalloc((void**)&gp_dev, size * sizeof(double));

    hipMemcpy(qold_dev, qold, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fm_dev, fm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fp_dev, fp, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gm_dev, gm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gp_dev, gp, size * sizeof(double), hipMemcpyHostToDevice);

    // this is not optimal
    dim3 dimGrid(mx, my, meqn);
    dim3 dimBlock(1, 1);
    cudaclaw_update_q_cuda<<<dimBlock, dimGrid>>>(mbc, dtdx, dtdy, 
                                                   qold_dev, fm_dev, fp_dev, 
                                                   gm_dev, gp_dev);
    //equivalent c loop
    /*
    int x_stride = meqn;
    int y_stride = (mx + 2 * mbc)*x_stride;
    for(int m=0;m<meqn;m++){
        for(int x=0;x<mx;x++){
            for(int y=0;y<my;y++){
                int i = m+(x+mbc)*x_stride+(y+mbc)*y_stride;
                qold[i] =qold[i] -dtdx * (fm[i+x_stride] - fp[i]) -
                                  dtdy * (gm[i+y_stride] - gp[i]);

            }
        }
    }
    */

	hipError_t code = hipPeekAtLastError();
    if(code!=hipSuccess){
        printf("ERROR: %s\n",hipGetErrorString(code));
    }

    hipMemcpy(qold, qold_dev, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(qold_dev);
    hipFree(fm_dev);
    hipFree(fp_dev);
    hipFree(gm_dev);
    hipFree(gp_dev);
}
#endif
