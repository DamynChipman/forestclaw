#include "hip/hip_runtime.h"
#include "cudaclaw_flux2.h"
#include <fclaw_base.h>  /* Needed for SC_MIN, SC_MAX */
#include "cudaclaw_allocate.h"

#include <math.h>


/* Use this version (in swirl example) to test performance hit with function
   pointers */
__device__ void rpn2adv(int idir, int meqn, int mwaves, 
                        int maux, double ql[], double qr[], 
                        double auxl[], double auxr[],
                        double wave[], double s[], 
                        double amdq[], double apdq[])
{
    /* Solve q_t + D q_x = 0, where D = diag([u,u,...,u]), D in R^{meqn x meqn} */
    int mq;

    for(mq = 0; mq < meqn; mq++)
    {
        wave[mq] = qr[mq] - ql[mq];        
    }

    s[0] = auxr[idir];    /* Assume all waves move at the same speed */

    for(mq = 0; mq < meqn; mq++)
    {
        amdq[mq] = SC_MIN(s[0], 0) * wave[mq];
        apdq[mq] = SC_MAX(s[0], 0) * wave[mq];            
    }
}

#define MEQN   32
#define MAUX   20 
#define MWAVES 10

extern "C"
{

__host__ int cudaclaw_check_dims(int meqn, int maux, int mwaves)
{
    int check;
    check = (meqn <= MEQN) && (maux <= MAUX) && (mwaves <= MWAVES);
    return check;
}
}


__global__
void cudaclaw_flux2_and_update_batch (int mx, int my, int meqn, int mbc, 
                                int maux, int mwaves, double dt,
                                cudaclaw_fluxes_t* array_fluxes_struct_dev,
                                cudaclaw_cuda_rpn2_t rpn2)
{
    // TODO: check this device function does not depend on blockIdx.z inside
    cudaclaw_flux2_and_update(mx,my,meqn,mbc,maux,mwaves,
            dt/array_fluxes_struct_dev[blockIdx.z].dx,
            dt/array_fluxes_struct_dev[blockIdx.z].dy,
                                    &(array_fluxes_struct_dev[blockIdx.z]),
                                    rpn2);
}

__device__  
void cudaclaw_flux2_and_update (int mx, int my, int meqn, int mbc, 
                           int maux, int mwaves, double dtdx, double dtdy,
                           cudaclaw_fluxes_t* fluxes,
                           cudaclaw_cuda_rpn2_t rpn2)
{
    //TODO: get this from Scott
}


__global__ void cudaclaw_flux2(int mx, int my, int meqn, int mbc,
                                int maux, int mwaves, 
                                double dtdx, double dtdy,
                                double* qold, double* aux, 
                                double* fm, double* fp, double* gm, double* gp,
                                double* waves, double *speeds,
                                cudaclaw_cuda_rpn2_t rpn2)
{
    int mq, mw, m;
    int xs, ys, zs;
    int I, I_q, I_aux, I_waves, I_speeds;

    /* Static memory seems much faster than dynamic memory */
    extern __shared__ double shared_mem[];
    double* ql   = shared_mem+threadIdx.x*(5*meqn+3*maux+mwaves+meqn*mwaves);//meqn
    double* qr   = ql+meqn;         //meqn
    double* qd   = qr+meqn;         //meqn
    double* auxl = qd+meqn;         //maux
    double* auxr = auxl+maux;       //maux
    double* auxd = auxr+maux;       //maux
    double* s    = auxd+maux;       //mwaves
    double* wave = s+mwaves;        //meqn*mwaves
    double* amdq = wave+meqn*mwaves;//meqn
    double* apdq = amdq+meqn;       //meqn

    int ifaces_x = mx+2*mbc-1;
    int ifaces_y = my+2*mbc-1;
    int num_ifaces = ifaces_x*ifaces_y;

    /* Compute strides */
    xs = 1;
    ys = (2*mbc + mx)*xs;
    zs = (2*mbc + my)*xs*ys;

    for(int thread_index = threadIdx.x; thread_index<num_ifaces; thread_index+=blockDim.x){

        int ix = thread_index%ifaces_x;
        int iy = thread_index/ifaces_y;

        /* (i,j) index */
        I = (iy + mbc-1)*ys + (ix + mbc-1)*xs;

        if (ix < mx + 2*mbc-1 && iy < my + 2*mbc-1)
        {
            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                ql[mq] = qold[I_q - xs];
                qr[mq] = qold[I_q];  
                qd[mq] = qold[I_q - ys];          
            }
            for(m = 0; m < maux; m++)
            {
                I_aux = I + m*zs;
                auxl[m] = aux[I_aux - xs];
                auxr[m] = aux[I_aux];
                auxd[m] = aux[I_aux - ys];
            }

            //rpn2adv(0, meqn, mwaves, maux, ql, qr, auxl, auxr, wave, s, amdq, apdq);
            rpn2(0, meqn, mwaves, maux, ql, qr, auxl, auxr, wave, s, amdq, apdq);

            /* Set value at left interface of cell I */
            for (mq = 0; mq < meqn; mq++) 
            {
                I_q = I + mq*zs;
                fp[I_q] = -apdq[mq]; 
                fm[I_q] = amdq[mq];
            }
#if 0        
            for (m = 0; m < meqn*mwaves; m++)
            {
                I_waves = I + m*zs;
                waves[I_waves] = wave[m];
            }
#endif        
            for (mw = 0; mw < mwaves; mw++)
            {
                I_speeds = I + mw*zs;
                speeds[I_speeds] = s[mw];
            } 


            rpn2(1, meqn, mwaves, maux, qd, qr, auxd, auxr, wave, s, amdq, apdq);

            /* Set value at bottom interface of cell I */
            for (mq = 0; mq < meqn; mq++) 
            {
                I_q = I + mq*zs;
                gp[I_q] = -apdq[mq]; 
                gm[I_q] = amdq[mq];
            }
            for (mw = 0; mw < mwaves; mw++)
            {
                I_speeds = I + (mwaves+mw)*zs;
                speeds[I_speeds] = s[mw];
            } 

#if 0        
            for (m = 0; m < meqn*mwaves; m++)
            {
                I_waves = I + m*zs;
                waves[I_waves] = wave[m];
            }
            for (mw = 0; mw < mwaves; mw++)
            {
                I_speeds = I + mw*zs;
                speeds[I_speeds] = s[mw];
            } 
#endif

        }
    }
    __syncthreads();
    for(int thread_index = threadIdx.x; thread_index<mx*my; thread_index+=blockDim.x){

        int ix = thread_index%mx;
        int iy = thread_index/my;

        I = (ix+mbc)*xs + (iy+mbc)*ys;

        for(mq = 0; mq < meqn; mq++)
        {
            I_q = I + mq*zs;
            qold[I_q] = qold[I_q] - dtdx * (fm[I_q + xs] - fp[I_q]) 
                                  - dtdy * (gm[I_q + ys] - gp[I_q]);
        }        

    }
}

__global__ void cudaclaw_compute_cfl(int idir, int mx, int my, int meqn, int mwaves, 
                                      int mbc, double dx, double dy, double dt, 
                                      double *speeds, double* cflgrid)
{
#if 0    
      # from fortran_source/cudaclaw_flux2.f */

c     # compute maximum wave speed for checking Courant number:
      cfl1d = 0.d0
      do 50 mw=1,mwaves
         do 50 i=1,mx+1
c          # if s>0 use dtdx1d(i) to compute CFL,
c          # if s<0 use dtdx1d(i-1) to compute CFL:
            cfl1d = dmax1(cfl1d, dtdx1d(i)*s(mw,i),
     &                          -dtdx1d(i-1)*s(mw,i))
   50       continue
#endif   
    /* Compute largest waves speeds, scaled by dt/dx,  on grid */


}


__device__ void cudaclaw_second_order(int idir, int mx, int my, int meqn, int mbc,
                                       int maux, double* qold, double* aux, double dx,
                                       double dy, double dt, double* cflgrid,
                                       double* fm, double* fp, double* gm, double* gp,
                                       double* waves, double *speeds,
                                       cudaclaw_cuda_rpn2_t rpn2, void* rpt2,
                                       int mwaves) 
{    
    int mq, mw, m;

    /* TODO : Limit waves here */


    /* TODO : Compute second order corrections */
    double dtdx = dt/dx;
    for(mq = 0; mq < meqn; mq++)
    {
        double cqxx = 0;
        for(mw = 0; mw < mwaves; mw++)
        {
            m = mw*meqn + mq;
            cqxx += fabs(speeds[mw])*(1.0 - fabs(speeds[mw])*dtdx)*waves[m];
        }
    }
}

