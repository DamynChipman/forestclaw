#include "hip/hip_runtime.h"
#include "tst_fptrs.h"
#include "user_f.h"

#include <stdio.h>


__global__ void kernel(fc2d_cuda_t f,float x, float *y)
{
    *y = f(x);
    return;
}

int main()
{
    float x;
    float y, *y_dev;
    fc2d_cuda_vt_t vt;

    //fc2d_assign_cuda_ptr_t f_assign_user;

    fc2d_cuda_t h_f;

    /* User definitions (in swirl_user, for example) */
    x = 5;
    assign_cuda_ptr2(&vt.h_f);


    /* Code */
    hipMalloc((void**) &y_dev, sizeof(float));

    kernel<<<1,1>>>(vt.h_f,x,y_dev);

    hipMemcpy(&y, y_dev, sizeof(float), hipMemcpyDeviceToHost);

    printf("x = %f; y = %f\n",x,y);

    return 0;
}

