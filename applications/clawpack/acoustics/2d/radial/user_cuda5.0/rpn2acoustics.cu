#include "../radial_user.h"

#include <fc2d_cudaclaw5.h>
#include <fclaw_base.h>  /* Needed for SC_MIN, SC_MAX */



__device__ void radial_rpn2acoustics(int idir, int meqn, int mwaves, 
                              int maux, double ql[], double qr[], 
                              double auxl[], double auxr[],
                              double wave[], double s[], 
                              double amdq[], double apdq[])
{
    /* wave[mwaves][meqn] */
    /* idir in 0,1 : needed to get correct  */

    // TODO: this should be replaced with acoustics riemann solver
    wave[0] = qr[0] - ql[0];
    s[0] = auxr[idir];
    amdq[0] = SC_MIN(auxr[idir], 0) * wave[0];
    apdq[0] = SC_MAX(auxr[idir], 0) * wave[0];
}

__device__ cudaclaw5_cuda_rpn2_t radial_rpn2 = radial_rpn2acoustics;

void radial_assign_rpn2(cudaclaw5_cuda_rpn2_t *rpn2)
{
    hipError_t ce = hipMemcpyFromSymbol(rpn2, HIP_SYMBOL(radial_rpn2), sizeof(cudaclaw5_cuda_rpn2_t));
    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (radial_rpn2adv): %s\n",hipGetErrorString(ce));
        exit(0);
    }    
}
